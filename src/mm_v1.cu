#include "hip/hip_runtime.h"
#include "mm_impl.cuh"

template <typename T>
__global__ void gemm_v01(
    size_t m,
    size_t n,
    size_t k,
    T alpha,
    T const *A,
    size_t lda,
    T const *B,
    size_t ldb,
    T beta,
    T *C,
    size_t ldc
){
    size_t col{blockIdx.x * blockDim.x + threadIdx.x}; // 连续变化
    size_t row{blockIdx.y * blockDim.y + threadIdx.y}; // 不变

    if (row < m && col < n)
    {
        T sum{static_cast<T>(0)};
        for (size_t i = 0; i < k; ++i)
            // A 不变, B 连续变化
            sum += A[row * lda + i] * B[i * ldb + col];
        // C 连续变化
        C[row * ldc + col] = alpha * sum + beta * C[row * ldc + col];
    }
}

template <typename T>
void launch_gemm_kernel_v01(
    size_t m,
    size_t n,
    size_t k,
    T const *alpha,
    T const *A,
    size_t lda,
    T const *B,
    size_t ldb,
    T const *beta,
    T *C,
    size_t ldc,
    hipStream_t stream
){
    dim3 const block_dim{BLOCK_SIZE, BLOCK_SIZE, 1};
    // TODO: static_cast<unsigned int>(m) 避免警告?
    dim3 const grid_dim{
        (static_cast<unsigned int>(n) + block_dim.x - 1) / block_dim.x,
        (static_cast<unsigned int>(m) + block_dim.y - 1) / block_dim.y,
    };
    gemm_v01<T><<<grid_dim, block_dim, 0U, stream>>>(
        m, n, k, *alpha, A, lda, B, ldb, *beta, C, ldc);
}

// Explicit instantiation.
template void launch_gemm_kernel_v01<float>(
    size_t m,
    size_t n,
    size_t k,
    float const *alpha,
    float const *A,
    size_t lda,
    float const *B,
    size_t ldb,
    float const *beta,
    float *C,
    size_t ldc,
    hipStream_t stream
);